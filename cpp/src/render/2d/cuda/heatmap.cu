#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2019-2020 Zilliz. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cmath>
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "render/2d/heatmap/set_color.h"
#include "render/utils/color/color_gradient.h"

namespace arctern {
namespace render {

unsigned int iDivUp(const unsigned int& a, const unsigned int& b) {
  return (a + b - 1) / b;
}

template <typename T>
__global__ void SetCountValue_gpu(float* out, uint32_t* in_x, uint32_t* in_y, T* in_c,
                                  int64_t num, int64_t width, int64_t height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  for (; i < num; i += blockDim.x * gridDim.x) {
    uint32_t vertice_x = in_x[i];
    uint32_t vertice_y = height - in_y[i] - 1;
    if (vertice_y > height || vertice_x > width) continue;
    if (vertice_y < 0 || vertice_x < 0) continue;
    int64_t index = vertice_y * width + vertice_x;
    if (index >= width * height) continue;
    out[index] += in_c[i];
  }
}

__global__ void HeatMapArray_gpu(float* in_count, float* out_count, float* kernel,
                                 int64_t kernel_size, int64_t width, int64_t height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int count_index = y * width + x;
  if (in_count[count_index] > 1e-8) {
    int r = kernel_size / 2;
    for (int m = -r; m <= r; m++) {
      if (x + m < 0 || x + m >= width) continue;
      for (int n = -r; n <= r; n++) {
        if (y + n < 0 || y + n >= height) continue;
        int kernel_index = (r + n) * (2 * r + 1) + (m + r);
        int dev_index = (y + n) * width + (x + m);
        out_count[dev_index] += in_count[count_index] * kernel[kernel_index];
      }
    }
  }
}

__global__ void MeanKernel_gpu(float* img_in, float* img_out, int64_t r, int64_t img_w,
                               int64_t img_h) {
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  double gradient = 0.0;
  if (r > 10) r = 10;
  int count = 0;
  if ((row >= 0) && (row < img_h) && (col >= 0) && (col < img_w)) {
    for (int m = -r; m <= r; m++) {
      if (row + m < 0 || row + m >= img_h) continue;
      for (int n = -r; n <= r; n++) {
        if (col + n < 0 || col + n >= img_w) continue;
        int y = row + m;
        int x = col + n;
        gradient += img_in[y * img_w + x];
        count++;
      }
    }
    img_out[row * img_w + col] = gradient / count;
  }
}

template <typename T>
void set_colors_gpu(float* colors, uint32_t* input_x, uint32_t* input_y, T* input_c,
                    int64_t num, VegaHeatMap& vega_heat_map) {
  WindowParams window_params = vega_heat_map.window_params();
  int64_t width = window_params.width();
  int64_t height = window_params.height();
  int64_t window_size = width * height;

  float* pix_count;
  uint32_t *in_x, *in_y;
  T* in_c;
  hipMalloc((void**)&pix_count, window_size * sizeof(float));
  hipMalloc((void**)&in_x, num * sizeof(uint32_t));
  hipMalloc((void**)&in_y, num * sizeof(uint32_t));
  hipMalloc((void**)&in_c, num * sizeof(T));
  hipMemset(pix_count, 0, window_size * sizeof(float));
  hipMemcpy(in_x, input_x, num * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(in_y, input_y, num * sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(in_c, input_c, num * sizeof(T), hipMemcpyHostToDevice);
  SetCountValue_gpu<T><<<256, 1024>>>(pix_count, in_x, in_y, in_c, num, width, height);

  double scale = vega_heat_map.map_scale() * 0.4;
  int d = pow(2, scale);
  int64_t kernel_size = d * 2 + 3;

  float* kernel = (float*)malloc(kernel_size * kernel_size * sizeof(float));
  guassiankernel2d(kernel, kernel_size, kernel_size, kernel_size, kernel_size);
  float* dev_kernel;
  hipMalloc((void**)&dev_kernel, kernel_size * kernel_size * sizeof(float));
  hipMemcpy(dev_kernel, kernel, kernel_size * kernel_size * sizeof(float),
             hipMemcpyHostToDevice);
  float* dev_count;
  hipMalloc((void**)&dev_count, window_size * sizeof(float));
  hipMemset(dev_count, 0, window_size * sizeof(float));

  const unsigned int blockW = 32;
  const unsigned int blockH = 32;
  const dim3 threadBlock(blockW, blockH);
  const dim3 grid(iDivUp(width, blockW), iDivUp(height, blockH));
  HeatMapArray_gpu<<<grid, threadBlock>>>(pix_count, dev_count, dev_kernel, kernel_size,
                                          width, height);

  float* color_count;
  hipMalloc((void**)&color_count, window_size * sizeof(float));
  hipMemset(color_count, 0, window_size * sizeof(float));
  int64_t mean_radius = (int)(log((kernel_size - 3) / 2) / 0.4);

  MeanKernel_gpu<<<grid, threadBlock>>>(dev_count, color_count, mean_radius + 1, width,
                                        height);
  MeanKernel_gpu<<<grid, threadBlock>>>(color_count, dev_count, mean_radius / 2 + 1,
                                        width, height);

  auto host_count = (float*)malloc(window_size * sizeof(float));
  hipMemcpy(host_count, dev_count, window_size * sizeof(float), hipMemcpyDeviceToHost);
  float max_pix = 0;
  for (auto k = 0; k < window_size; k++) {
    if (max_pix < host_count[k]) max_pix = host_count[k];
  }
  ColorGradient color_gradient;
  color_gradient.createDefaultHeatMapGradient();

  int64_t c_offset = 0;
  for (auto j = 0; j < window_size; j++) {
    float value = host_count[j] / max_pix;
    float color_r, color_g, color_b;
    color_gradient.getColorAtValue(value, color_r, color_g, color_b);
    colors[c_offset++] = color_r;
    colors[c_offset++] = color_g;
    colors[c_offset++] = color_b;
    colors[c_offset++] = value;
  }

  free(kernel);
  free(host_count);
  hipFree(pix_count);
  hipFree(dev_kernel);
  hipFree(dev_count);
  hipFree(color_count);
  hipFree(in_x);
  hipFree(in_y);
  hipFree(in_c);
}

}  // namespace render
}  // namespace arctern

#define TEMPLATE_GEN_PREFIX
#define T int8_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T int16_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T int32_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T int64_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T uint8_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T uint16_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T uint32_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T uint64_t
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T float
#include "render/2d/heatmap/set_color.inl"  // NOLINT

#define T double
#include "render/2d/heatmap/set_color.inl"  // NOLINT
