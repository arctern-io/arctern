#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

//
// Created by mike on 2/10/20.
//
#include <hip/hip_runtime.h>

#include <cmath>

#include "gis/cuda/common/gpu_memory.h"
#include "gis/cuda/functor/st_distance.h"

namespace zilliz {
namespace gis {
namespace cuda {
namespace {
inline DEVICE_RUNNABLE double Point2PointDistance(ConstGpuContext& left,
                                                  ConstGpuContext& right, int index) {
  auto lv = left.get_value_ptr(index);
  auto rv = right.get_value_ptr(index);
  auto dx = (lv[0] - rv[0]);
  auto dy = (lv[1] - rv[1]);
  return sqrt(dx * dx + dy * dy);
}

__global__ void ST_DistanceKernel(ConstGpuContext left, ConstGpuContext right,
                                  double* result) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < left.size) {
    auto left_tag = left.get_tag(tid);
    auto right_tag = right.get_tag(tid);
    // handle 2d case only for now
    assert(left_tag.get_space_type() == WkbSpaceType::XY);
    assert(right_tag.get_space_type() == WkbSpaceType::XY);
    // handle point to point case only
    if (left_tag.get_category() == WkbCategory::kPoint &&
        right_tag.get_category() == WkbCategory::kPoint) {
      result[tid] = Point2PointDistance(left, right, tid);
    } else {
      result[tid] = NAN;
    }
  }
}
}  // namespace

void ST_Distance(const GeometryVector& left_vec, const GeometryVector& right_vec,
                 double* host_results) {
  assert(left_vec.size() == right_vec.size());
  auto left_ctx_holder = left_vec.CreateReadGpuContext();
  auto right_ctx_holder = right_vec.CreateReadGpuContext();
  auto dev_result = GpuMakeUniqueArray<double>(left_vec.size());
  {
    auto config = GetKernelExecConfig(left_vec.size());
    ST_DistanceKernel<<<config.grid_dim, config.block_dim>>>(
        *left_ctx_holder, *right_ctx_holder, dev_result.get());
  }
  GpuMemcpy(host_results, dev_result.get(), left_vec.size());
}

}  // namespace cuda
}  // namespace gis
}  // namespace zilliz
